#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <vector>

#define IMG_W 28
#define IMG_H 28

#define IMG_C 60000
#define IMG_C_TEST 10000

#define IMG_SIZE (IMG_W * IMG_H)

char pixelToChar(uint8_t pixel) {
    const char* levels = " .:-=+*#%@";
    int index = pixel * 10 / 256;
    return levels[index];
}

uint32_t readBigEndianUInt32(FILE* fp) {
    uint8_t bytes[4];
    if (fread(bytes, 1, 4, fp) != 4) {
        fprintf(stderr, "Failed to read 4 bytes\n");
        return 0;
    }
    return (bytes[0] << 24) | (bytes[1] << 16) | (bytes[2] << 8) | bytes[3];
}

uint8_t *readDataset(const char *dataset_filepath, uint32_t &num_images) {
    FILE *fp = fopen(dataset_filepath, "rb");
    uint8_t *data = nullptr;

    if (!fp) goto error;

    {
        uint32_t magic_number = readBigEndianUInt32(fp);
        num_images            = readBigEndianUInt32(fp);
        uint32_t num_rows     = readBigEndianUInt32(fp);
        uint32_t num_cols     = readBigEndianUInt32(fp);

        if (magic_number != 2051) goto error;
        if (num_cols != IMG_W) goto error;
        if (num_rows != IMG_H) goto error;
    }

    data = new uint8_t[num_images * IMG_SIZE];
    if (fread(data, 1, num_images * IMG_SIZE, fp) != num_images * IMG_SIZE) goto error;

    fclose(fp);

    return data;

error:

    perror("Error reading dataset");
    if (fp) fclose(fp);
    if (data) delete[] data;
    return nullptr;
}

uint8_t *readLabels(const char *labels_filepath, uint32_t &num_labels) {
    FILE *fp = fopen(labels_filepath, "rb");
    uint8_t *labels = nullptr;

    if (!fp) goto error;

    {
        uint32_t magic_number = readBigEndianUInt32(fp);
        num_labels            = readBigEndianUInt32(fp);

        if (magic_number != 2049) goto error;
    }

    labels = new uint8_t[num_labels];
    if (fread(labels, 1, num_labels, fp) != num_labels) goto error;

    fclose(fp);
    return labels;

error:

    perror("Error reading labels");
    if (fp) fclose(fp);
    if (labels) delete[] labels;
    return nullptr;
}

__global__ void computeDistances(const uint8_t *dataset, const uint8_t *query, uint32_t *distances) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < IMG_C) {
        uint32_t sum = 0.0f;
        for (int i = 0; i < IMG_SIZE; i++) {
            uint32_t diff = dataset[idx * IMG_SIZE + i] - query[i];
            sum += diff * diff;
        }
        distances[idx] = sum;
    }
}

int findMinIndex(const uint32_t *distances, int size) {
    uint32_t min_val = UINT32_MAX;
    int min_idx = -1;
    for (int i = 0; i < size; i++) {
        if (distances[i] < min_val) {
            min_val = distances[i];
            min_idx = i;
        }
    }
    return min_idx;
}

int main(int argc, const char **argv) {
    uint32_t num_images_dataset = 0;
    uint8_t *dataset = readDataset("data/train-images-idx3-ubyte", num_images_dataset);
    if (!dataset || num_images_dataset != IMG_C) return 1;

    uint32_t num_labels_dataset = 0;
    uint8_t *labels = readLabels("data/train-labels-idx1-ubyte", num_labels_dataset);
    if (!labels || num_labels_dataset != IMG_C) return 2;

    uint32_t num_images_test = 0;
    uint8_t *dataset_test = readDataset("data/t10k-images-idx3-ubyte", num_images_test);
    if (!dataset_test || num_images_test != 10000) return 3;
    
    uint32_t num_labels_test = 0;
    uint8_t *labels_test = readLabels("data/t10k-labels-idx1-ubyte", num_labels_test);
    if (!labels_test || num_labels_test != IMG_C_TEST) return 4;

    uint8_t *d_dataset, *d_query;
    uint32_t *d_distances;
    hipMalloc(&d_dataset, IMG_C * IMG_SIZE);
    hipMalloc(&d_query, IMG_SIZE);
    hipMalloc(&d_distances, IMG_C * sizeof(uint32_t));

    hipMemcpy(d_dataset, dataset, IMG_C * IMG_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_query, dataset_test, IMG_SIZE, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (IMG_C + threadsPerBlock - 1) / threadsPerBlock;

    computeDistances<<<blocksPerGrid, threadsPerBlock>>>(d_dataset, d_query, d_distances);

    uint32_t *distances = new uint32_t[IMG_C];
    hipMemcpy(distances, d_distances, IMG_C * sizeof(uint32_t), hipMemcpyDeviceToHost);

    int nearestIdx = findMinIndex(distances, IMG_C);
    printf("Guess: %d\n", labels[nearestIdx]);
    printf("Actual: %d\n", labels_test[0]);

    return 0;
}